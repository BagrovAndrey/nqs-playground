#include "dotu.hpp"
#include <ATen/cuda/HIPContext.h>

TCM_NAMESPACE_BEGIN
namespace gpu {

TCM_EXPORT auto dotu_gpu(TensorInfo<std::complex<float> const> const& x,
                         TensorInfo<std::complex<float> const> const& y,
                         c10::Device const device) -> std::complex<double>
{
    std::complex<float> result;
    hipSetDevice(device.index());
    auto const handle = at::cuda::getCurrentCUDABlasHandle();
    auto const status = hipblasCdotu(handle, x.size(),
                                    reinterpret_cast<hipComplex const*>(x.data), x.stride(),
                                    reinterpret_cast<hipComplex const*>(y.data), y.stride(),
                                    reinterpret_cast<hipComplex*>(&result));
    TCM_CHECK(status == HIPBLAS_STATUS_SUCCESS, std::runtime_error, "CUBLAS Error");
    return result;
}

} // namespace gpu
TCM_NAMESPACE_END

